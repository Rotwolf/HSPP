
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <random>
#include <chrono>
#include <fstream>
#include <sstream>
#include <utility>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

__global__ void setup_kernel(hiprandState *state, int N, unsigned long long seed){

    int ameisenid = blockIdx.x;
    if (threadIdx.x == 0) {
        hiprand_init(seed, ameisenid, 0, &state[ameisenid]);     // inizialisiert in jedem Block einen eigenen Zufallszahlengenerator
    }
}

__global__ void tour_konstruktions_kernel(
    hiprandState *my_curandstate, 
    int N,          // Anzahl der Ameisen insgesamt
    int cldim,      // größe der TSP-Instanz
    float alpha,     
    float beta,     
    int *cost, 
    float *phero,
    int *d_route    // array für Routenrückgabe
    ) {

    extern __shared__ int s[];   // Shared variables inizialisierung
    int* route = s;
    float* probabilities = (float*)&route[cldim];
    bool* visited = (bool*)&probabilities[cldim];
    __shared__ float sum;

    int ameisenid = blockIdx.x; 
    int stadtid = threadIdx.x;

    if (stadtid < cldim) {
        if (stadtid == 0) {
            for (int i = 0; i < cldim; i++) visited[i] = false;  // visited wieder mit false inizialisieren
            float myrandstart = hiprand_uniform(my_curandstate+ameisenid); // bestimmen der Ausgangsstadt
            myrandstart *= (cldim -1 +0.99999); // its very important that this is not even one 9 longer, since above .999999 the Computer would roud up, leading to an Error
            int start = (int)truncf(myrandstart);
            route[0] = start;
            visited[start] = true;
        }
        __syncthreads();
        for (int i = 1; i < cldim-1; i++) { //alle anderen Städte bis auf die letzte bestimmen.
            int aktuellestadt = route[i-1];
            if (!visited[stadtid]) { //Wahrscheinlichkietsberechnung nur bei sädten die noch nicht besucht wurden
                probabilities[stadtid] = __powf(phero[aktuellestadt*cldim+stadtid]+0.1E-28, alpha) * __powf(1./(cost[aktuellestadt*cldim+stadtid]+0.1E-3), beta);
            } else {
                probabilities[stadtid] = 0;
            }
            __syncthreads();
            if (stadtid == 0) {
                sum = 0;
                for (int j = 0; j < cldim; j++) {
                    sum += probabilities[j];        // summe bilden
                }
            }
            __syncthreads();
            probabilities[stadtid] /= sum;  // durch summe Teilen
            __syncthreads();
            if (stadtid == 0) {    // roulett wheel selection
                float r = hiprand_uniform(my_curandstate+ameisenid); // r in Menge (0,1]
                float sum_prob = 0;
                int next = -1;
                for (int j = 0; j < cldim; j++) {
                    sum_prob += probabilities[j];
                    if (r <= sum_prob) {
                        next = j;
                        break;
                    }
                }
                if (next == -1 && r > sum_prob) {       // prüft, ob das Problem die ungenauigkeit der Errechneten Wahrscheinlichkeiten ist.
                    int j = cldim-1;
                    while (visited[j]) j--;
                    next = j;                       // setzen auf deie letzte nciht besuchte Stadt
                }
                route[i] = next;
                visited[next] = true;
            }
            __syncthreads();
        }

        if (stadtid == 0) {
            int i = 0;
            while (visited[i]) i++;
            route[cldim-1] = i;         // letzte Stadt per Ausschluss bestimmen
            for (int i = 0; i < cldim; i++) d_route[ameisenid*cldim+i] = route[i]; // Route übertragen
        }
    }
}

__global__ void pheromon_evaporation_kernel(
    float p,
    float *phero
    ) {

    int kante = blockIdx.x * blockDim.x + threadIdx.x; 
    phero[kante] = phero[kante] * (1-p);

}

__global__ void pheromon_aktualisierungs_kernel( 
    int cldim,
    int *lenlist,
    int *cost, 
    float *phero,
    int *route
    ) {
    extern __shared__ int partial_len[];
    __shared__ float pherodelta;

    int ameisenid = blockIdx.x;
    int schrittid = threadIdx.x;
    int stadt_i, stadt_j; 
    if (schrittid == 0) { // die einzellnen Threads suchen nur den weg von einem zum anderen Knoten heraus 
        stadt_i = route[ameisenid*cldim + cldim - 1];
        stadt_j = route[ameisenid*cldim];
    } else {
        stadt_i = route[ameisenid*cldim + schrittid - 1];
        stadt_j = route[ameisenid*cldim + schrittid];
    }

    partial_len[schrittid] = cost[stadt_i * cldim + stadt_j];
    __syncthreads();
    if (schrittid == 0) { // ein einzelner Thread im Block addiet es zur Tourlaenge zusammen und gibt den wert zurück
        int len = 0;
        for (int i = 0; i < cldim; i++) len += partial_len[i];
        //write it down, analyse it later sequentialy.
        lenlist[ameisenid] = len; 
        pherodelta = 1./len;
    }
    __syncthreads();
    atomicAdd(&phero[stadt_i * cldim + stadt_j], pherodelta); // hinzufügen der Pheromone
    atomicAdd(&phero[stadt_j * cldim + stadt_i], pherodelta);
}

vector<pair<float, float>> parseTSPFile(const string& filenameshort) {
    vector<pair<float, float>> coordinates;

    string filename = "C:/Users/jonie/Desktop/hausarbeit/testtsps/" + filenameshort +".tsp"; //hier lokalen weg zu den tsp instanzen einfügen.
    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Error opening file: " << filename << endl;
        return coordinates;
    }

    string line;
    // Skip header lines
    bool sectionFound = false;
    while (getline(file, line)) {
        if (line == "NODE_COORD_SECTION") {
            sectionFound = true;
            break;
        }
    }

    if (!sectionFound) {
        cerr << "NODE_COORD_SECTION not found in file: " << filename << endl;
        return coordinates;
    }

    // Read node coordinates
    int index;
    float x, y;
    while (file >> index >> x >> y) {
        coordinates.push_back(make_pair(x, y));
    }

    file.close();
    return coordinates;
}

class ac {
    private:
        int *cost;     // Kosten im CPU
        int *d_cost;   // Kosten im GPU
        float *phero;
        float *d_phero;
        int *d_route;
        int *route;
        int *d_lenlist;
        int *lenlist;
        vector<pair<float, float>> cl;
        int cldim;
        float alpha;
        float beta;
        vector<int> vbestwaysofar;
        int lenofbestwaysofar;
        int lenofbestway;
        bool solisopt;
        random_device rd;
        mt19937 gen;
        hiprandState *d_state;
        int N;
        int block_size;
        int blocks;
        unsigned long long seed;
        void inizialisiereACO(vector<pair<float, float>> citylist, int lenofbestroute, int anzAnts) {
            cl = citylist;
            cldim = citylist.size();
            alpha = 1;          // festlegung der werte alpha und Beta
            beta = 2;
            lenofbestwaysofar = 0;              //
            lenofbestway = lenofbestroute;      // werden nur benötigt wenn versucht werden soll eine Länge zu unterschreiten, z.B. die optimale
            solisopt = false;                   //
            N = anzAnts;
            seed = time(NULL);
            // Es folgen die allocierungen und inizialisierungen
            cost = (int *)malloc((cldim*cldim)*sizeof(int));
            for (int x = 0; x < cldim; x++) {
                for (int y = x+1; y < cldim; y++) {
                    float way = round(sqrt(pow(cl[x].first - cl[y].first, 2) + pow(cl[x].second - cl[y].second, 2)));
                    cost[x*cldim+y] = way;
                    cost[y*cldim+x] = way;
                }
            }
            for (int x = 0; x < cldim; x++) cost[x*cldim+x] = 0;
            phero = (float *)malloc((cldim*cldim)*sizeof(float));
            for (int x = 0; x < cldim*cldim; x++) phero[x] = 0;
            lenlist = (int *)malloc(N*sizeof(int));
            route = (int *)malloc(N*cldim*sizeof(int));
            for (int i = 0; i < cldim; i++) {
                vbestwaysofar.push_back(i);
            }
            lenofbestwaysofar = calulate_way_from_route(vbestwaysofar);
        }
        void initialisiereGPU() {  // cudaMallocs, cudaCopy und setupCernel
            hipMalloc(&d_state, N*sizeof(hiprandState));
            block_size = cldim; 
            blocks = N; 
            setup_kernel<<<blocks,block_size>>>(d_state, N, seed);

            hipMalloc((void **) &d_cost, cldim*cldim*sizeof(int));
            hipMemcpy(d_cost, cost, cldim*cldim*sizeof(int), hipMemcpyHostToDevice);
            hipMalloc((void **) &d_phero, cldim*cldim*sizeof(float));
            hipMemcpy(d_phero, phero, cldim*cldim*sizeof(float), hipMemcpyHostToDevice);
            hipMalloc((void **) &d_lenlist, N*sizeof(int));
            hipMalloc((void **) &d_route, N*cldim*sizeof(int));
        }
        void oneIteration(float p) {
            //kernel call
            tour_konstruktions_kernel<<<blocks, block_size, cldim*sizeof(int)+cldim*sizeof(float)+cldim*sizeof(bool)>>>(d_state, N, cldim, alpha, beta, d_cost, d_phero, d_route);
            hipDeviceSynchronize();
            pheromon_evaporation_kernel<<<cldim, cldim>>>(p,d_phero);
            hipDeviceSynchronize(); //?
            pheromon_aktualisierungs_kernel<<<blocks, block_size, cldim*sizeof(int)>>>(cldim, d_lenlist, d_cost, d_phero, d_route);
            hipMemcpy(lenlist, d_lenlist, N*sizeof(int), hipMemcpyDeviceToHost);

            int min = 0;
            for (int i = 1; i < N; i++) if (lenlist[i] < lenlist[min]) min = i; // sucht kleinste länge

            if (lenlist[min] < lenofbestwaysofar){      // wenn die kleinste länge kleiner als die momentan Kleiste Tour ist wird diese überschrieben
                hipMemcpy(route, d_route, N*cldim*sizeof(int), hipMemcpyDeviceToHost);
                vbestwaysofar.clear();
                for (int i = 0; i < cldim; i++) {
                    vbestwaysofar.push_back(route[min * cldim + i]);
                }
                lenofbestwaysofar = lenlist[min];
                if (lenofbestwaysofar <= lenofbestway) {
                    solisopt = true; 
                }
            }
        }
        int calulate_way_from_route(vector<int> route) { // utility
            int way = 0;
            for (int i = 0; i < cldim-1; i++) {
                way += cost[route[i]*cldim+route[i+1]];
            }
            way += cost[route[cldim-1]*cldim+route[0]];
            return way;
        }
    public:
        ac(vector<pair<float, float>> citylist, float lenofbestroute=0, int anzAnts=2048) : gen(rd()) { //constructor
            inizialisiereACO(citylist, lenofbestroute, anzAnts);
            initialisiereGPU();
        }
        void doIteration(float p=0.5) {
            oneIteration(p);
        }
        vector<vector<int>> getcost() {
            vector<vector<int>> resultmatrix (cldim, vector<int>(cldim));
            for (int i = 0; i < cldim; i++) {
                for (int j = 0; j < cldim; j++) {
                    resultmatrix[i][j] = cost[i * cldim + j];
                }
            }
            return resultmatrix;
        }
        vector<vector<float>> getphero() {
            vector<vector<float>> resultmatrix (cldim, vector<float>(cldim));
            for (int i = 0; i < cldim; i++) {
                for (int j = 0; j < cldim; j++) {
                    resultmatrix[i][j] = phero[i * cldim + j];
                }
            }
            return resultmatrix;
        }
        vector<int> getbestroute() {
            return vbestwaysofar;
        }
        int getbestroutelen() {
            return lenofbestwaysofar;
        }
        bool issolopt() {
            return solisopt;
        }
        void freeall(void) {
            free(cost);
            free(phero);
            free(lenlist);
            hipFree(d_state);
            hipFree(d_cost);
            hipFree(d_phero);
            hipFree(d_route);
            hipFree(d_lenlist);
        }
};

int main(void) {
      
    vector<pair<float, float>> dj38 = parseTSPFile("dj38");
    int soldj38 = 6656;

    vector<pair<float, float>> lu980 = parseTSPFile("lu980");
    int sollu980 = 11340;

    vector<pair<float, float>> qa194 = parseTSPFile("qa194");
    int solqa194 = 9352;

    vector<pair<float, float>> a280 = parseTSPFile("a280");
    int sola280 = 2579;
    
    vector<pair<float, float>> d198 = parseTSPFile("d198");
    int sold198 = 15780;

    vector<pair<float, float>> lin318 = parseTSPFile("lin318");
    int sollin318 = 42029;

    vector<pair<float, float>> pcb442 = parseTSPFile("pcb442");
    int solpcb442 = 50778;

    vector<pair<float, float>> pr1002 = parseTSPFile("pr1002");
    int solpr1002 = 259045;

    vector<pair<float, float>> rat783 = parseTSPFile("rat783");
    int solrat783 = 8806;

    vector<int> coloniesize = {1024, 2048, 4096, 8192}; 

/*
    // Berechnung der Durchschnittslänge
    for (int i = 0; i < coloniesize.size(); i++) {

        int anzberechungen = 30;
        vector<int> mlength;
        mlength.resize(anzberechungen);

        for (int k = 0; k < anzberechungen; k++) { 

            int anziter = 30;
            vector<pair<float, float>> citylits = lin318;
            float p = 0.5;

            int bestroutlen = INT_MAX;
            ac region(citylits, 0, coloniesize[i]);

            for (int j = 0; j < anziter; j++) {
                region.doIteration(p);      
            }
            bestroutlen = region.getbestroutelen(); 
            //cout << "bestroutelen: " << bestroutlen << endl;
            mlength[k] = bestroutlen;
            region.freeall();
      
        }

        float summe = 0.0;
        for (int j = 0; j < anzberechungen; j++) {
            summe += mlength[j];
        }
        float avg = summe / mlength.size();
        cout << "Die Durchschnittliche routenlaenge fuer " << coloniesize[i] << " betraegt: " << avg << endl;
    }

    return 0;
*/

    // Berechnung der Durchschnittszeit
    for (int i = 0; i < coloniesize.size(); i++) {

        int anzberechungen = 30;
        vector<pair<float, float>> citylits = d198;
        float p = 0.5;
        vector<chrono::duration<float>> listofdurations;
        listofdurations.resize(anzberechungen);

        vector<int> bestrout;
        int bestroutlen = INT_MAX;
        ac region(citylits, 0, coloniesize[i]);

        for (int j = 0; j < anzberechungen; j++) {
            auto start = chrono::high_resolution_clock::now();

            region.doIteration(p);

            auto end = chrono::high_resolution_clock::now();
            listofdurations[j] = end - start;

            bestroutlen = region.getbestroutelen();
            bestrout = region.getbestroute();
            
        }
        
        cout << "bestroutelen: " << bestroutlen << endl;

        // bestrout = region.getbestroute();
        // cout << "bestroute: [";
        // for (const auto& element : bestrout) {
        //     cout << element << ", ";
        // }
        // cout << endl;

        region.freeall();

        float summe = 0.0;
        for (int j = 0; j < anzberechungen; j++) {
            summe += listofdurations[j].count();
        }

        float avg = summe / listofdurations.size();
        cout << "Die Durchschnittliche Ausfuehrungszeit fuer " << coloniesize[i] << " betraegt: " << avg << " Sekunden." << endl;
        // cout << "suration values: ";
        // for (const auto& element : listofdurations) {
        //     cout << element.count() << " ";
        // }
        // cout << endl;
    }

    return 0;
}
