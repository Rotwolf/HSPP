
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void) {
        hipDeviceProp_t deviceProp;
        if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) {
                printf("Get device properties failed.\n");
                return 1;
        } else {
                printf("The warp size is %d.\n", deviceProp.warpSize);
                return 0;
        }
}