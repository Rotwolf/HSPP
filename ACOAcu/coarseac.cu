#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <random>
#include <chrono>
#include <fstream>
#include <sstream>
#include <utility>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

__global__ void setup_kernel(hiprandState *state, int N){

    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    for (int j = idx; j<N; j += blockDim.x * gridDim.x) {
        hiprand_init(4444, j, 0, &state[j]);
    }
}

__global__ void generate_kernel(
    hiprandState *my_curandstate, 
    int N, 
    int cldim, 
    double alpha,
    double beta,
    double *cost, 
    double *phero,
    int *d_route
    ) {

    int idx = threadIdx.x + blockDim.x*blockIdx.x;
  
    for (int j = idx; j<N; j += blockDim.x * gridDim.x) {
        int *route = (int *)malloc(cldim*sizeof(int));
        bool* visited = new bool[cldim]();
        for (int i = 0; i < cldim; i++) visited[i] = false;
        double myrandstart = hiprand_uniform_double(my_curandstate+idx);
        myrandstart *= (cldim -1 +0.99999999);
        int start = (int)truncf(myrandstart);
        route[0] = start;
        visited[start] = true;
        for (int i = 1; i < cldim-1; i++) {
            int current = route[i-1];
            double sum = 0;
            double* probabilities = new double[cldim]();
            for (int j = 0; j < cldim; j++) {
                if (!visited[j]) {
                    probabilities[j] = pow(phero[current*cldim+j]+0.1E-200, alpha) * pow(1/cost[current*cldim+j], beta);
                    sum += probabilities[j];
                }
            }
            for (int j = 0; j < cldim; j++) {
                if (!visited[j]) {
                    probabilities[j] /= sum;
                    /*if (isnan(probabilities[j])) {
                        cout << "it is nan: " << endl; 
                        cout << "phero: " << phero[current][j] << endl;
                        cout << "cost: " << cost[current][j] << endl;
                        cout << "sum: " << sum << endl;
                    }*/
                }
            }
            double r = hiprand_uniform_double(my_curandstate+idx);
            //For (0,1]
            double sum_prob = 0;
            int next = -1;
            for (int j = 0; j < cldim; j++) {
                sum_prob += probabilities[j];
                if (r <= sum_prob) {
                    next = j;
                    break;
                }
            }
            /*//For [0,1)
            double sum_prob = 1;
            int next = -1;
            for (int j = 0; j < cldim; j++) {
                sum_prob -= probabilities[j];
                if (r >= sum_prob) {
                    next = j;
                    break;
                }
            }
            */
            //if (next == -1) { 
                //add error msg? 
            //}
            free(probabilities);

            route[i] = next;
            visited[next] = true;
        }
        int i = 0;
        while (visited[i]) i++;
        route[cldim-1] = i;
        visited[i] = true;

        for (int i = 0; i < cldim; i++) d_route[idx*cldim+i] = route[i];

        free(route);
        free(visited);
    }
}

vector<pair<double, double>> parseTSPFile(const string& filenameshort) {
    vector<pair<double, double>> coordinates;

    string filename = "C:/Users/jonie/Desktop/hausarbeit/testtsps/" + filenameshort +".tsp";
    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Error opening file: " << filename << endl;
        return coordinates;
    }

    string line;
    // Skip header lines
    bool sectionFound = false;
    while (getline(file, line)) {
        if (line == "NODE_COORD_SECTION") {
            sectionFound = true;
            break;
        }
    }

    if (!sectionFound) {
        cerr << "NODE_COORD_SECTION not found in file: " << filename << endl;
        return coordinates;
    }

    // Read node coordinates
    int index;
    double x, y;
    while (file >> index >> x >> y) {
        coordinates.push_back(make_pair(x, y));
    }

    file.close();
    return coordinates;
}

class ac {
    private:
        double *cost;
        double *d_cost;
        double *phero;
        double *d_phero;
        double *nextphero;
        int *d_route;
        vector<pair<double, double>> cl;
        int cldim;
        double alpha;
        double beta;
        vector<int> bestwaysofar;
        double lenofbestwaysofar;
        double lenofbestway;
        bool solisopt;
        random_device rd;
        mt19937 gen;
        hiprandState *d_state;
        int N;
        int block_size;
        int blocks;
        void generateMatrixes() {
            cost = (double *)malloc((cldim*cldim)*sizeof(double));
            for (int x = 0; x < cldim; x++) {
                for (int y = x+1; y < cldim; y++) {
                    double way = round(sqrt(pow(cl[x].first - cl[y].first, 2) + pow(cl[x].second - cl[y].second, 2)));
                    cost[x*cldim+y] = way;
                    cost[y*cldim+x] = way;
                }
            }
            for (int x = 0; x < cldim; x++) cost[x*cldim+x] = 0;
            phero = (double *)malloc((cldim*cldim)*sizeof(double));
            for (int x = 0; x < cldim; x++) {
                for (int y = x+1; y < cldim; y++) { // prüfen ob starten komplett mit 0 etwas ändert
                    phero[x*cldim+y] = 1;
                    phero[y*cldim+x] = 1;
                }
            }
            for (int x = 0; x < cldim; x++) phero[x*cldim+x] = 0;
            nextphero = (double *)malloc((cldim*cldim)*sizeof(double));
            for (int x = 0; x < cldim*cldim; x++) nextphero[x] = 0;
            for (int i = 0; i < cldim; i++) {
                bestwaysofar.push_back(i);
            }
            lenofbestwaysofar = calulate_way_from_route(bestwaysofar);
        }
        void setupCuda() {
            hipMalloc(&d_state, N*sizeof(hiprandState));
            block_size = 32;
            blocks = (N / block_size)+1; // without +1 nothing works????????????
            setup_kernel<<<blocks,block_size>>>(d_state, N);

            hipMalloc((void **) &d_cost, cldim*cldim*sizeof(double));
            hipMemcpy(d_cost, cost, cldim*cldim*sizeof(double), hipMemcpyHostToDevice);
            hipMalloc((void **) &d_phero, cldim*cldim*sizeof(double));
            hipMalloc((void **) &d_route, N*cldim*sizeof(int));
            
        }
        void tourconstruction() {
            
            int *route = (int *)malloc((N*cldim)*sizeof(int));

            hipMemcpy(d_phero, phero, cldim*cldim*sizeof(double), hipMemcpyHostToDevice);
            //kernel call
            generate_kernel<<<blocks,block_size>>>(d_state, N, cldim, alpha, beta, d_cost, d_phero, d_route);
            hipMemcpy(route, d_route, (N*cldim)*sizeof(int), hipMemcpyDeviceToHost);

            for (int j = 0; j < N; j++) {
                vector<int> vroute;
                for (int i = 0; i < cldim; i++) {
                    vroute.push_back(route[j * cldim + i]);
                }
                double len = calulate_way_from_route(vroute);
                if (len < lenofbestwaysofar) {
                    bestwaysofar = vroute;
                    lenofbestwaysofar = len;
                    if (lenofbestwaysofar == lenofbestway) {
                        solisopt = true;
                        return;
                    }
                }
                double nlen = 1/len;
                for (int i = 0; i < cldim-1; i++) {
                    nextphero[vroute[i]*cldim+vroute[i+1]] += nlen;
                    nextphero[vroute[i+1]*cldim+vroute[i]] += nlen;
                }
                nextphero[vroute[cldim-1]*cldim+vroute[0]] += nlen;
                nextphero[vroute[0]*cldim+vroute[cldim-1]] += nlen;    
            }
            free(route);
        }
        void phermoneupdate(double p) {
            for (int i = 0; i < cldim; i++) {
                for (int j = 0; j < cldim; j++) {
                    phero[i*cldim+j] = (1-p) * phero[i*cldim+j] + nextphero[i*cldim+j];
                    nextphero[i*cldim+j] = 0;
                }
            }
        }
        double calulate_way_from_route(vector<int> route) {
            double way = 0;
            for (int i = 0; i < cldim-1; i++) {
                way += cost[route[i]*cldim+route[i+1]];
            }
            way += cost[route[cldim-1]*cldim+route[0]];
            return way;
        }
    public:
        ac(vector<pair<double, double>> citylist, double lenofbestroute=0, int anzAnts=2000) : gen(rd()) {
            cl = citylist;
            cldim = citylist.size();
            alpha = 1;
            beta = 1;
            lenofbestwaysofar = 0;
            lenofbestway = lenofbestroute;
            solisopt = false;
            N = anzAnts;

            generateMatrixes();
            setupCuda();
        }
        void doIteration(double p=0.5) {
            tourconstruction();
            if (solisopt) return;
            phermoneupdate(p);
        }
        vector<vector<double>> getcost() {
            vector<vector<double>> resultmatrix (cldim, vector<double>(cldim));
            for (int i = 0; i < cldim; i++) {
                for (int j = 0; j < cldim; j++) {
                    resultmatrix[i][j] = cost[i * cldim + j];
                }
            }
            return resultmatrix;
        }
        vector<vector<double>> getphero() {
            vector<vector<double>> resultmatrix (cldim, vector<double>(cldim));
            for (int i = 0; i < cldim; i++) {
                for (int j = 0; j < cldim; j++) {
                    resultmatrix[i][j] = phero[i * cldim + j];
                }
            }
            return resultmatrix;
        }
        vector<int> getbestroute() {
            return bestwaysofar;
        }
        double getbestroutelen() {
            return lenofbestwaysofar;
        }
        bool issolopt() {
            return solisopt;
        }
        void freeCuda(void) {
            hipFree(d_state);
            hipFree(d_cost);
        }
};

int main(void) {
    vector<pair<double, double>> cl1 = {{182,663},{232,33},{230,787},{370,676},{256,996},{600,247},{33,672},{119,225},{525,985},{716,397}}; //(3, 8, 4, 2, 0, 6, 7, 1, 5, 9) or [1, 7, 6, 0, 2, 4, 8, 3, 9, 5]
    
    vector<pair<double, double>> dj38 = parseTSPFile("dj38");//{{11003.611100, 42102.500000},{11108.611100, 42373.888900},{11133.333300, 42885.833300},{11155.833300, 42712.500000},{11183.333300, 42933.333300},{11297.500000, 42853.333300},{11310.277800, 42929.444400},{11416.666700, 42983.333300},{11423.888900, 43000.277800},{11438.333300, 42057.222200},{11461.111100, 43252.777800},{11485.555600, 43187.222200},{11503.055600, 42855.277800},{11511.388900, 42106.388900},{11522.222200, 42841.944400},{11569.444400, 43136.666700},{11583.333300, 43150.000000},{11595.000000, 43148.055600},{11600.000000, 43150.000000},{11690.555600, 42686.666700},{11715.833300, 41836.111100}, {11751.111100, 42814.444400},{11770.277800, 42651.944400},{11785.277800, 42884.444400},{11822.777800, 42673.611100},{11846.944400, 42660.555600},{11963.055600, 43290.555600},{11973.055600, 43026.111100},{12058.333300, 42195.555600},{12149.444400, 42477.500000},{12286.944400, 43355.555600},{12300.000000, 42433.333300},{12355.833300, 43156.388900},{12363.333300, 43189.166700},{12372.777800, 42711.388900},{12386.666700, 43334.722200},{12421.666700, 42895.555600},{12645.000000, 42973.333300}};// (12 14 19 22 24 25 21 23 27 26 30 35 33 32 36 34 31 29 28 20 13 9 0 0 1 3 2 4 5 6 7 8 10 11 15 16 17 18)
    double soldj38 = 6656;

    vector<pair<double, double>> lu980 = parseTSPFile("lu980");;
    double sollu980 = 11340;

    vector<pair<double, double>> qa194 = parseTSPFile("qa194");;
    double solqa194 = 9352;

    vector<pair<double, double>> a280 = parseTSPFile("a280");//{{288, 149}, {288, 129}, {270, 133}, {256, 141}, {256, 157}, {246, 157}, {236, 169}, {228, 169}, {228, 161}, {220, 169}, {212, 169}, {204, 169}, {196, 169}, {188, 169}, {196, 161}, {188, 145}, {172, 145}, {164, 145}, {156, 145}, {148, 145}, {140, 145}, {148, 169}, {164, 169}, {172, 169}, {156, 169}, {140, 169}, {132, 169}, {124, 169}, {116, 161}, {104, 153}, {104, 161}, {104, 169}, {90, 165}, {80, 157}, {64, 157}, {64, 165}, {56, 169}, {56, 161}, {56, 153}, {56, 145}, {56, 137}, {56, 129}, {56, 121}, {40, 121}, {40, 129}, {40, 137}, {40, 145}, {40, 153}, {40, 161}, {40, 169}, {32, 169}, {32, 161}, {32, 153}, {32, 145}, {32, 137}, {32, 129}, {32, 121}, {32, 113}, {40, 113}, {56, 113}, {56, 105}, {48, 99}, {40, 99}, {32, 97}, {32, 89}, {24, 89}, {16, 97}, {16, 109}, {8, 109}, {8, 97}, {8, 89}, {8, 81}, {8, 73}, {8, 65}, {8, 57}, {16, 57}, {8, 49}, {8, 41}, {24, 45}, {32, 41}, {32, 49}, {32, 57}, {32, 65}, {32, 73}, {32, 81}, {40, 83}, {40, 73}, {40, 63}, {40, 51}, {44, 43}, {44, 35}, {44, 27}, {32, 25}, {24, 25}, {16, 25}, {16, 17}, {24, 17}, {32, 17}, {44, 11}, {56, 9}, {56, 17}, {56, 25}, {56, 33}, {56, 41}, {64, 41}, {72, 41}, {72, 49}, {56, 49}, {48, 51}, {56, 57}, {56, 65}, {48, 63}, {48, 73}, {56, 73}, {56, 81}, {48, 83}, {56, 89}, {56, 97}, {104, 97}, {104, 105}, {104, 113}, {104, 121}, {104, 129}, {104, 137}, {104, 145}, {116, 145}, {124, 145}, {132, 145}, {132, 137}, {140, 137}, {148, 137}, {156, 137}, {164, 137}, {172, 125}, {172, 117}, {172, 109}, {172, 101}, {172, 93}, {172, 85}, {180, 85}, {180, 77}, {180, 69}, {180, 61}, {180, 53}, {172, 53}, {172, 61}, {172, 69}, {172, 77}, {164, 81}, {148, 85}, {124, 85}, {124, 93}, {124, 109}, {124, 125}, {124, 117}, {124, 101}, {104, 89}, {104, 81}, {104, 73}, {104, 65}, {104, 49}, {104, 41}, {104, 33}, {104, 25}, {104, 17}, {92, 9}, {80, 9}, {72, 9}, {64, 21}, {72, 25}, {80, 25}, {80, 41}, {88, 49}, {104, 57}, {124, 69}, {124, 77}, {132, 81}, {140, 65}, {132, 61}, {124, 61}, {124, 53}, {124, 45}, {124, 37}, {124, 29}, {132, 21}, {124, 21}, {120, 9}, {128, 9}, {136, 9}, {148, 9}, {162, 9}, {156, 25}, {172, 21}, {180, 21}, {180, 29}, {172, 29}, {172, 37}, {172, 45}, {180, 45}, {180, 37}, {188, 41}, {196, 49}, {204, 57}, {212, 65}, {220, 73}, {228, 69}, {228, 77}, {236, 77}, {236, 69}, {236, 61}, {228, 61}, {228, 53}, {236, 53}, {236, 45}, {228, 45}, {228, 37}, {236, 37}, {236, 29}, {228, 29}, {228, 21}, {236, 21}, {252, 21}, {260, 29}, {260, 37}, {260, 45}, {260, 53}, {260, 61}, {260, 69}, {260, 77}, {276, 77}, {276, 69}, {276, 61}, {276, 53}, {284, 53}, {284, 61}, {284, 69}, {284, 77}, {284, 85}, {284, 93}, {284, 101}, {288, 109}, {280, 109}, {276, 101}, {276, 93}, {276, 85}, {268, 97}, {260, 109}, {252, 101}, {260, 93}, {260, 85}, {236, 85}, {228, 85}, {228, 93}, {236, 93}, {236, 101}, {228, 101}, {228, 109}, {228, 117}, {228, 125}, {220, 125}, {212, 117}, {204, 109}, {196, 101}, {188, 93}, {180, 93}, {180, 101}, {180, 109}, {180, 117}, {180, 125}, {196, 145}, {204, 145}, {212, 145}, {220, 145}, {228, 145}, {236, 145}, {246, 141}, {252, 125}, {260, 129}, {280, 133}};
    double sola280 = 2579;
    
    vector<pair<double, double>> d198 = parseTSPFile("d198");;
    double sold198 = 15780;

    vector<pair<double, double>> lin318 = parseTSPFile("lin318");;
    double sollin318 = 42029;

    vector<pair<double, double>> pcb442 = parseTSPFile("pcb442");;
    double solpcb442 = 50778;

    vector<pair<double, double>> pr1002 = parseTSPFile("pr1002");;
    double solpr1002 = 259045;

    vector<pair<double, double>> rat783 = parseTSPFile("rat783");;
    double solrat783 = 8806;

    vector<int> bestrout;
    int bestroutlen = INT_MAX;
    int newbestroutlen;
    int lastbestroutechange = 0;
    ac region(qa194, solqa194, 2000);

    auto start = chrono::high_resolution_clock::now();

    int i = -1;
    while (!region.issolopt() && lastbestroutechange<500) {
        i++;
        cout <<  i << endl; 
        newbestroutlen = region.getbestroutelen();
        cout << "bestroutlen: " << newbestroutlen << endl;
        if (newbestroutlen < bestroutlen) {
            bestroutlen = newbestroutlen;
            lastbestroutechange = 0;
        } else {
            lastbestroutechange++;
        }
        cout << "lastchange was: * " << lastbestroutechange << " * Iterations ago." << endl;
        bestrout = region.getbestroute();
        cout << "bestroute: [";
        for (const auto& element : bestrout) {
            cout << element << ", ";
        }
        cout << endl;
        region.doIteration(0.5);
    }

    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double> duration = end - start;
    cout << "Die Ausfuehrungszeit betraegt: " << duration.count() << " Sekunden." << endl;
    bestroutlen = region.getbestroutelen();
    cout << "bestroutelen: " << bestroutlen << endl;
    bestrout = region.getbestroute();
    cout << "bestroute: [";
    for (const auto& element : bestrout) {
        cout << element << ", ";
    }
    cout << endl;
    cout << "sorted bestroute: ";
    sort(bestrout.begin(), bestrout.end());
    for (const auto& element : bestrout) {
        cout << element << " ";
    }
    cout << endl;

    region.freeCuda();

    return 0;
}

